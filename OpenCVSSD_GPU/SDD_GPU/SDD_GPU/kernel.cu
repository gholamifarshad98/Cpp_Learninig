
#include <iostream>
#include <string>
#include <sstream>
#include <iomanip>
#include <stdexcept>
#include <opencv2/core/utility.hpp>
#include "opencv2/core/cuda.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/imgproc.hpp"

using namespace cv;
using namespace std;




int main(int argc, char** argv)
{

	bool running;
	Mat left_src, right_src;
	Mat left, right;
	cuda::GpuMat d_left, d_right;

	int ndisp = 88;

	Ptr<cuda::StereoBM> bm;

	bm = cuda::createStereoBM(ndisp);



	// Load images
	left_src = imread("s1.png");
	right_src = imread("s2.png");

	cvtColor(left_src, left, COLOR_BGR2GRAY);
	cvtColor(right_src, right, COLOR_BGR2GRAY);


	d_left.upload(left);
	d_right.upload(right);

	imshow("left", left);
	imshow("right", right);



	// Prepare disparity map of specified type
	Mat disp(left.size(), CV_8U);
	cuda::GpuMat d_disp(left.size(), CV_8U);

	cout << endl;


	running = true;
	while (running)
	{

		bm->compute(d_left, d_right, d_disp);

		// Show results
		d_disp.download(disp);

		imshow("disparity", (Mat_<uchar>)disp);

		waitKey(1);
	}

	return 0;
}





















/*

int main(void) {
	// Allocate & initialize host data - run on the host
	Mat leftImage = imread("view0.png", 0);
	Mat rightImage = imread("view1.png", 0);
	if (leftImage.empty() || rightImage.empty()) {
		cout << "Error in reading Left or right image" << endl;
	}
	int ro;
	int co;
	co = leftImage.cols;
	ro = leftImage.rows;
	Mat Disparity;
	leftImage.at<uchar>(ro - 5, co - 9) = 255;
	imshow("left Image", leftImage);
	imshow("right Image", rightImage);
	cout << "Width is ===>" << co << " and Hight is ====> " << ro << endl;




	waitKey(0);



	// host copies of a, b, c
	Mat *d_leftImage, *d_rightImage, *d_result; // device copies of a, b, c
	// Allocate space for device copies of a, b, c
	int size = sizeof(d_leftImage);
	hipMalloc((void **)&d_leftImage, size);
	hipMalloc((void **)&d_leftImage, size);
	hipMalloc((void **)&d_result, size);

	// Copy a & b from the host to the device
	hipMemcpy(d_leftImage, &leftImage, size, hipMemcpyHostToDevice);
	hipMemcpy(d_rightImage, &rightImage, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	add <<< 1, 1 >>>(d_a, d_b, d_c);

	// Copy result back to the host
	hipMemcpy(&Disparity, d_result, size, hipMemcpyDeviceToHost);
	// Cleanup
	hipFree(d_leftImage); hipFree(d_rightImage); hipFree(d_result);

	
	return 0;
}
*/
Mat MySSD(Mat &left, Mat &right, int win, int MaxOffset, int ro, int co) {
	int halfWin = int(win / 2);

	Mat result(ro, co, CV_8UC1, Scalar(255));

	for (int i = halfWin; i < ro - halfWin; ++i) {
		for (int j = halfWin; j < co - halfWin; ++j) {
			int ssd_reserved = 9999635;
			int bestOffset = 0;
			double offset_adjust = 255 / MaxOffset;
			for (int offset = 0; offset <= MaxOffset; ++offset) {
				int ssd = 0;
				int ssdTemp = 0;
				for (int u = -halfWin; u <= halfWin; u++) {
					for (int v = -halfWin; v <= halfWin; v++) {
						ssdTemp = (left.at<uchar>((i + u), (j + v)) - right.at<uchar>((i + u), (j + v - offset)));
						ssdTemp = ssdTemp*ssdTemp;
						ssd = ssd + ssdTemp;
					}
				}
				if (ssd < ssd_reserved) {
					bestOffset = offset;
					ssd_reserved = ssd;

				}
			}
			result.at<uchar>(i, j) = bestOffset* offset_adjust;
		}
	}
	return result;
}



